#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add(float* A, float* B, float* C, int n)
{
	int i = threadIdx.x;

	if (i < n) C[i] = A[i] + B[i];
}


void print_cuda_errors(hipError_t status)
{
	if (status != hipSuccess)
	{
		std::cout << "error" << hipGetErrorString(status) << std::endl;
	}
}



int main()
{
	size_t size = 4 * sizeof(float);

	float a[] = {0, 1, 2, 3};
	float b[] = {0, 1, 2, 3};

	float* c;

	print_cuda_errors(hipHostMalloc(&c, size, hipHostMallocDefault));
	
	float* d_a;
	float* d_b;
	float* d_c;

	print_cuda_errors(hipMalloc(&d_a, size));
	print_cuda_errors(hipMalloc(&d_b, size));
	print_cuda_errors(hipMalloc(&d_c, size));
	
	print_cuda_errors(hipMemcpy(d_a, a, size, hipMemcpyHostToDevice));
	print_cuda_errors(hipMemcpy(d_b, b, size, hipMemcpyHostToDevice));

	int tPB = 256;
	int bPG = (4 + tPB - 1) / tPB;

	add<<<1, 64>>>(d_a, d_b, d_c, 4);

	print_cuda_errors(hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost));

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	for (int i = 0; i < 4; i++)
	{
		std::cout << c[i] << " ";
	}
	std::cout << std::endl;

	hipFree(c);

	return 0;
}
